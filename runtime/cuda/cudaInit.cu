#include <jni.h>
#include <hip/hip_runtime.h>

extern "C" JNIEXPORT void JNICALL Java_ppl_delite_runtime_executor_GPUExecutionThread_initializeDevice(JNIEnv* env, jobject obj, jint deviceNum);

JNIEXPORT void JNICALL Java_ppl_delite_runtime_executor_GPUExecutionThread_initializeDevice(JNIEnv* env, jobject obj, jint deviceNum) {
	hipSetDevice(deviceNum); //chose device num
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync); //set device options
	hipDeviceSynchronize();//called to initialize the device (can take a while)
}
