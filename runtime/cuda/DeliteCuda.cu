#include <hip/hip_runtime.h>
#include <list>
#include <map>
#include <queue>
#include <iostream>

using namespace std;

extern hipStream_t h2dStream;
extern hipStream_t d2hStream;

void* lastValue;

struct FreeItem {
	hipEvent_t event;
	list<void*>* keys;
};

queue<FreeItem>* freeList = new queue<FreeItem>();

map<void*,void*>* cudaMemoryMap = new map<void*,void*>();


void DeliteCudaMalloc(void** ptr, int size) {
	size_t free;
	size_t total;
	hipMemGetInfo(&free, &total);
	while (free < size) {
		if (freeList->size() == 0) {
			cout << "Insufficient device memory" << endl;
			exit(-1);
		}
		FreeItem item = freeList->front();
        	freeList->pop();
		while (hipEventQuery(item.event) != hipSuccess)
			hipEventSynchronize(item.event);
		list<void*>::iterator iter;
		for (iter = item.keys->begin(); iter != item.keys->end(); iter++) {
			hipFree(cudaMemoryMap->find(*iter)->second);
		}
		hipMemGetInfo(&free, &total);
	}
	hipMalloc(ptr, size);
	lastValue = *ptr;
}

/* void DeliteCudaMalloc(void** ptr, int size) {
        while (freeList->size() > 0) {
		FreeItem item = freeList->front();
 	        freeList->pop();
		while (cudaEventQuery(item.event) != cudaSuccess)
			cudaEventSynchronize(item.event);
		list<void*>::iterator iter;
		for (iter = item.keys->begin(); iter != item.keys->end(); iter++) {
			void* freePtr = cudaMemoryMap->find(*iter)->second;
			if (cudaSuccess != cudaFree(freePtr))
				cout << "bad free pointer" << endl;
			else
				cout << "freed successfully: " << (long) freePtr << endl;
		}
	}
	if (cudaSuccess != cudaMalloc(ptr, size))
		cout << "malloc failed" << endl;
	else
		cout << "allocated successfully: " << (long) *ptr << endl;
	lastValue = *ptr;
} */

void DeliteCudaMallocHost(void **ptr, int size) {
	hipHostAlloc(ptr, size, hipHostMallocDefault);
}

void DeliteCudaMemcpyHtoDAsync(void* dptr, void* sptr, int size) {
	hipMemcpyAsync(dptr, sptr, size, hipMemcpyHostToDevice, h2dStream);
}

void DeliteCudaMemcpyDtoHAsync(void* dptr, void* sptr, int size) {
	hipMemcpyAsync(dptr, sptr, size, hipMemcpyDeviceToHost, d2hStream);
	hipStreamSynchronize(d2hStream);
}
