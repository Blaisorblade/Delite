#include <hip/hip_runtime.h>

extern hipStream_t h2dStream;
extern hipStream_t d2hStream;

void DeliteCudaMallocHost(void **ptr, int size) {
	hipHostAlloc(ptr, size, hipHostMallocDefault);
}

void DeliteCudaMalloc(void **ptr, int size) {
	hipMalloc(ptr, size);
}

void DeliteCudaMemcpyHtoDAsync(void *dptr, void *sptr, int size) {
	hipMemcpyAsync(dptr, sptr, size, hipMemcpyHostToDevice, h2dStream);

}

void DeliteCudaMemcpyDtoHAsync(void *dptr, void *sptr, int size) {
	hipMemcpyAsync(dptr, sptr, size, hipMemcpyDeviceToHost, d2hStream);
	hipStreamSynchronize(d2hStream);
}
